#include "hip/hip_runtime.h"
/*
  This file is part of Leela Chess Zero.
  Copyright (C) 2018-2019 The LCZero Authors

  Leela Chess is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  Leela Chess is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with Leela Chess.  If not, see <http://www.gnu.org/licenses/>.

  Additional permission under GNU GPL version 3 section 7

  If you modify this Program, or any covered work, by linking or
  combining it with NVIDIA Corporation's libraries from the NVIDIA CUDA
  Toolkit and the NVIDIA CUDA Deep Neural Network library (or a
  modified version of those libraries), containing parts covered by the
  terms of the respective license agreement, the licensors of this
  Program grant you additional permission to convey the resulting work.
*/

#include <cassert>

#include "cuda_common.h"
#include "winograd_helper.inc"

namespace lczero {
namespace cudnn_backend {
namespace {
constexpr int kInputPlanes = 112;
}  // namespace

/////////////////////////////////////////////////////////////////////////////
//          Simple CUDA kernels used by certain layers                     //
/////////////////////////////////////////////////////////////////////////////

template <typename T>
__global__ void addVectors_kernel(T* c, T* a, T* b, int size, int asize,
                                  int bsize, ActivationFunction activation) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < size) {
    float aVal = 0;
    float bVal = 0;
    if (a) aVal = (float)(a[i % asize]);
    if (b) bVal = (float)(b[i % bsize]);

    float cVal = aVal + bVal;

    cVal = activate(cVal, activation);

    c[i] = (T)cVal;
  }
}

// Adds two vectors (possibly of different sizes), also do optional relu
// activation.
template <typename T>
void addVectors(T* c, T* a, T* b, int size, int asize, int bsize,
                ActivationFunction activation, hipStream_t stream) {
  const int kBlockSize = 256;
  int blocks = DivUp(size, kBlockSize);

  addVectors_kernel<<<blocks, kBlockSize, 0, stream>>>(c, a, b, size, asize,
                                                       bsize, activation);
  ReportCUDAErrors(hipGetLastError());
}

template <typename T>
__global__ void addBias_NCHW_kernel(T* c, T* a, T* b, int N, int C, int H,
                                    int W, ActivationFunction activation) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int size = N * C * H * W;
  if (i < size) {
    float aVal = (float)a[i];

    // All this math can be optimized, but the kernel is memory bound anyway.
    int biasIndex = (i / (H * W)) % C;
    float bVal = (float)b[biasIndex];

    float cVal = aVal + bVal;

    cVal = activate(cVal, activation);

    c[i] = (T)cVal;
  }
}

// Add bias to convolution's output.
template <typename T>
void addBias_NCHW(T* c, T* a, T* b, int N, int C, int H, int W,
                  ActivationFunction activation, hipStream_t stream) {
  int size = N * C * H * W;
  const int kBlockSize = 256;
  int blocks = DivUp(size, kBlockSize);

  addBias_NCHW_kernel<<<blocks, kBlockSize, 0, stream>>>(c, a, b, N, C, H, W,
                                                         activation);
  ReportCUDAErrors(hipGetLastError());
}

template <typename dT, typename sT>
__device__ dT readNCHW(const sT* input_tensor, int n, int c, int h, int w,
                       int Nin, int Cin, int H, int W) {
  if (n >= Nin || c >= Cin) return 0;

  int index;
  index = n;
  index *= Cin;
  index += c;
  index *= H;
  index += h;
  index *= W;
  index += w;

  return (dT)(input_tensor[index]);
}

template <typename dT, typename sT>
__global__ void NCHWtoNHWC_kernel(dT* output_tensor, const sT* input_tensor,
                                  int Nin, int Cin, int Nout, int Cout, int H,
                                  int W) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= Nout * Cout * H * W) return;

  int index = tid;

  int c = (index % Cout);
  index /= Cout;
  int w = index % W;
  index /= W;
  int h = index % H;
  index /= H;
  int n = index;

  output_tensor[tid] =
      readNCHW<dT, sT>(input_tensor, n, c, h, w, Nin, Cin, H, W);
}

template <typename DstType, typename SrcType>
void convertNCHWtoNHWC(DstType* output_tensor, const SrcType* input_tensor,
                       int Nin, int Cin, int Nout, int Cout, int H, int W) {
  size_t numElements = Nout * Cout * H * W;
  const int blockSize = 256;
  int blocks = DivUp(numElements, blockSize);
  NCHWtoNHWC_kernel<<<blocks, blockSize>>>(output_tensor, input_tensor, Nin,
                                           Cin, Nout, Cout, H, W);
}

template <typename DstType, typename SrcType>
__global__ void copyTypeConverted_kernel(DstType* op, SrcType* ip, int N) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= N) return;

  DstType el = (DstType)ip[tid];
  op[tid] = el;
}

template <typename DstType, typename SrcType>
void copyTypeConverted(DstType* op, SrcType* ip, int N, hipStream_t stream) {
  const int kBlockSize = 256;
  int blocks = DivUp(N, kBlockSize);
  copyTypeConverted_kernel<<<blocks, kBlockSize, 0, stream>>>(op, ip, N);
}

template <typename T>
__global__ void batchNorm_kernel(T* output, const T* input, const T* skipInput,
                                 int N, int C, int H, int W, const float* means,
                                 const float* varMultipliers,
                                 ActivationFunction activation) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;

  int wIndex = 0;
  if (sizeof(T) == sizeof(float))
    wIndex = (index / (H * W)) % C;  // NCHW for fp32.
  else
    wIndex = index % C;  // NHWC for fp16.

  float el = input[index];
  float mean = means[wIndex];
  float varMulti = varMultipliers[wIndex];

  el -= mean;
  el *= varMulti;

  if (skipInput) el += (float)skipInput[index];

  el = activate(el, activation);

  output[index] = (T)el;
}

// Every thread processes single element.
template <typename T>
void batchNorm(T* output, const T* input, const T* skipInput, int N, int C,
               int H, int W, float* means, float* var_multipliers,
               ActivationFunction activation) {
  const int total_elements = N * C * H * W;
  const int kBlockSize = 256;
  int blocks = DivUp(total_elements, kBlockSize);

  batchNorm_kernel<<<blocks, kBlockSize>>>(output, input, skipInput, N, C, H, W,
                                           means, var_multipliers, activation);

  ReportCUDAErrors(hipGetLastError());
}

__global__ void expandPlanes_kernel_Fp32_NCHW(float* output,
                                              const uint64_t* masks,
                                              const float* values, int n) {
  // Block size of 256, same mask/val for 64 consecutive threads.
  constexpr int kNumShmemElements = 256 / 64;

  __shared__ uint64_t shMasks[kNumShmemElements];
  __shared__ float shVals[kNumShmemElements];

  int index = threadIdx.x + blockDim.x * blockIdx.x;

  int planeIndex = index >> 6;

  if (planeIndex >= n) return;

  // Load inputs to shared memory.
  if (threadIdx.x < kNumShmemElements) {
    shMasks[threadIdx.x] = masks[planeIndex + threadIdx.x];
    shVals[threadIdx.x] = values[planeIndex + threadIdx.x];
  }
  __syncthreads();

  uint64_t mask = shMasks[threadIdx.x >> 6];

  int sqIndex = index & 0x3F;
  float op = 0;

  bool set = !!(mask & (1ull << sqIndex));
  if (set) {
    op = shVals[threadIdx.x >> 6];
  }
  output[index] = op;
}

void expandPlanes_Fp32_NCHW(float* output, const uint64_t* masks,
                            const float* values, int n, hipStream_t stream) {
  int threads = n * 8 * 8;  // Each thread writes a single element.
  const int blockSize = 256;
  int blocks = DivUp(threads, blockSize);
  expandPlanes_kernel_Fp32_NCHW<<<blocks, blockSize, 0, stream>>>(output, masks,
                                                                  values, n);
  ReportCUDAErrors(hipGetLastError());
}

// TODO: Can optimize using shared memory if this becomes a bottleneck.
__global__ void expandPlanes_kernel_Fp16_NHWC(half* output,
                                              const uint64_t* masks,
                                              const float* values, int n) {
  const int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index >= n * 8 * 8) return;

  const int planeIndex = index % kInputPlanes;
  const int boardIndex = index / (kInputPlanes * 8 * 8);
  const int sqIndex = (index / kInputPlanes) & 0x3F;

  uint64_t mask = masks[boardIndex * kInputPlanes + planeIndex];

  half op = 0;
  bool set = !!(mask & (1ull << sqIndex));
  if (set) {
    float val = values[boardIndex * kInputPlanes + planeIndex];
    op = (half)val;
  }
  output[index] = op;
}

void expandPlanes_Fp16_NHWC(half* output, const uint64_t* masks,
                            const float* values, int n, hipStream_t stream) {
  int threads = n * 8 * 8;  // Each thread writes a single element.
  const int kBlockSize = 256;
  int blocks = DivUp(threads, kBlockSize);
  expandPlanes_kernel_Fp16_NHWC<<<blocks, kBlockSize, 0, stream>>>(
      output, masks, values, n);
  ReportCUDAErrors(hipGetLastError());
}

__global__ void expandPlanes_kernel_Fp16_NCHW(half* output,
                                              const uint64_t* masks,
                                              const float* values, int n) {
  // block size of 256, same mask/val for 64 consecutive threads
  constexpr int kNumShmemElements = 256 / 64;

  __shared__ uint64_t shMasks[kNumShmemElements];
  __shared__ half shVals[kNumShmemElements];

  int index = threadIdx.x + blockDim.x * blockIdx.x;

  int planeIndex = index >> 6;

  if (planeIndex >= n) return;

  // load inputs to shared memory
  if (threadIdx.x < kNumShmemElements) {
    shMasks[threadIdx.x] = masks[planeIndex + threadIdx.x];
    shVals[threadIdx.x] = values[planeIndex + threadIdx.x];
  }
  __syncthreads();

  uint64_t mask = shMasks[threadIdx.x >> 6];

  int sqIndex = index & 0x3F;
  half op = 0;

  bool set = !!(mask & (1ull << sqIndex));
  if (set) {
    op = (half)shVals[threadIdx.x >> 6];
  }
  output[index] = op;
}

void expandPlanes_Fp16_NCHW(half* output, const uint64_t* masks,
                            const float* values, int n, hipStream_t stream) {
  int threads = n * 8 * 8;  // each thread writes a single element
  const int blockSize = 256;
  int blocks = DivUp(threads, blockSize);
  expandPlanes_kernel_Fp16_NCHW<<<blocks, blockSize, 0, stream>>>(output, masks,
                                                                  values, n);
  ReportCUDAErrors(hipGetLastError());
}

template <typename T>
__global__ void globalScale_kernel(T* output, const T* input,
                                   const T* scaleBias, const T* prevLayerBias,
                                   int inputSize, int C,
                                   ActivationFunction activation) {
  const int kPlaneSize = 64;

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid > inputSize) return;

  int nc = tid / kPlaneSize;
  int n = nc / C;
  int c = nc % C;

  float val1 = input[tid];   // Output of residual block to be scaled.
  float val2 = output[tid];  // Skip connection to be added directly.

  if (prevLayerBias) {
    val1 += (float)(prevLayerBias[c]);
  }

  int startIdx = n * 2 * C;  // Scale and bias interleaved.

  float s = scaleBias[startIdx + c];
  s = 1.0f / (1.0f + exp(-s));  // Sigmoid on scale.

  float b = scaleBias[startIdx + c + C];

  float op = val1 * s + val2 + b;
  op = activate(op, activation);
  output[tid] = (T)op;
}

__global__ void globalScale_kernel_fp16_nhwc(half* output, const half* input,
                                             const half* scaleBias,
                                             const half* prevLayerBias,
                                             int inputSize, int C, int HWC,
                                             ActivationFunction activation) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid > inputSize) return;

  int c = tid % C;
  int n = tid / (HWC);

  float val1 = (float)input[tid];   // Output of residual block to be scaled.
  float val2 = (float)output[tid];  // Skip connection to be added directly.
  if (prevLayerBias) {
    val1 += (float)prevLayerBias[c];
  }

  int startIdx = n * 2 * C;  // Scale and bias interleaved.

  float s = scaleBias[startIdx + c];
  s = 1.0f / (1.0f + exp(-s));  // Sigmoid on scale.

  float b = scaleBias[startIdx + c + C];

  float op = val1 * s + val2 + b;
  op = activate(op, activation);

  output[tid] = (half)op;
}

// N blocks.
// C threads per block.
// 'HWC' input data processed by thread block.
// Each thread writes a single output.
__global__ void globalAvgPool_kernel_NHWC_fp16(half* output, const half* input,
                                               const half* prevLayerBias,
                                               int inputSize, int outputSize) {
  const int elementsPerThread = 64;  // 8x8 board.

  int blockStart = blockIdx.x * blockDim.x;

  float S = 0;

#pragma unroll
  for (int i = 0; i < elementsPerThread; i++) {
    int localIndex = i * blockDim.x + threadIdx.x;
    int inputIndex = blockStart * elementsPerThread + localIndex;
    if (inputIndex < inputSize) S += (float)(input[inputIndex]);
  }

  float avg = S / elementsPerThread;

  // Add bias from previous layer.
  if (prevLayerBias) avg += (float)(prevLayerBias[threadIdx.x]);

  int opIndex = blockStart + threadIdx.x;
  if (opIndex < outputSize) output[opIndex] = (half)avg;
}

// Each thread reads 2 inputs (8x8/32), and each warp writes a single output.
template <typename T>
__global__ void globalAvgPool_kernel(T* output, const T* input,
                                     const T* prevLayerBias, int inputSize,
                                     int outputSize, int C) {
  const int elementsPerWarp = 64;
  const int elementsPerThread = 2;

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  int laneId = threadIdx.x & 0x1F;
  int laneStartIndex = (tid - laneId) * elementsPerThread;

  // Compute per-thread sum for elementsPerThread elements.
  float S = 0;

#pragma unroll
  for (int i = 0; i < elementsPerWarp; i += 32) {
    int index = laneStartIndex + laneId + i;
    if (index < inputSize) S += (float)(input[index]);
  }

// Compute warp wide sum (for entire plane - elementsPerWarp elements).
#pragma unroll
  for (int offset = 1; offset < 32; offset *= 2) {
    S += __shfl_down_sync(0xFFFFFFFF, S, offset);
  }

  float avg = S / elementsPerWarp;
  int opIndex = tid >> 5;

  // First thread in warp has the sum, write it in output.
  if (laneId == 0) {
    if (opIndex < outputSize) {
      if (prevLayerBias) avg += (float)prevLayerBias[opIndex % C];
      output[opIndex] = (T)avg;
    }
  }
}

template <typename T>
void globalAvgPool(int N, int C, T* output, const T* input,
                   const T* prevLayerBias, bool nhwc) {
  const int kPlaneSize = 64;

  const bool fp16 = std::is_same<half, T>::value;
  if (nhwc) {
    assert(fp16);
    // For NHWC fp16, simply launch N blocks, each with C threads.
    globalAvgPool_kernel_NHWC_fp16<<<N, C>>>((half*)output, (half*)input,
                                             (half*)prevLayerBias,
                                             N * C * kPlaneSize, N * C);
  } else {
    // For NCHW layout (used with fp32),
    // each warp processes a full plane (64 elements), and writes a single
    // average N*C warps are launched.

    const int kTotalWarps = N * C;
    const int kWarpsPerBlock = 8;
    const int kBlockSize = kWarpsPerBlock * 32;

    int blocks = DivUp(kTotalWarps, kWarpsPerBlock);
    globalAvgPool_kernel<<<blocks, kBlockSize>>>(output, input, prevLayerBias,
                                                 N * C * kPlaneSize, N * C, C);
  }
  ReportCUDAErrors(hipGetLastError());
}

template <typename T>
void globalScale(int N, int C, T* output, const T* input, const T* scaleBias,
                 const T* prevLayerBias, bool nhwc,
                 ActivationFunction activation) {
  const bool fp16 = std::is_same<half, T>::value;

  // Each thread writes one output.
  const int kBlockSize = 256;
  const int kBlocks = DivUp(N * 8 * 8 * C, kBlockSize);

  if (nhwc) {
    assert(fp16);
    globalScale_kernel_fp16_nhwc<<<kBlocks, kBlockSize>>>(
        (half*)output, (half*)input, (half*)scaleBias, (half*)prevLayerBias,
        N * C * 8 * 8, C, 8 * 8 * C, activation);
  } else {
    globalScale_kernel<<<kBlocks, kBlockSize>>>(
        output, input, scaleBias, prevLayerBias, N * C * 8 * 8, C, activation);
  }
  ReportCUDAErrors(hipGetLastError());
}

template <typename T>
__global__ void policyMap_kernel(T* output, const T* input,
                                 const short* indices, int N, int inputSize,
                                 int usedSize, int outputSize) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  int n = tid / usedSize;
  int i = tid % usedSize;

  if (n >= N) return;

  int j = indices[i];

  if (j >= 0) {
    output[n * outputSize + j] = input[n * inputSize + i];
  }
}

template <typename T>
void PolicyMap(int N, T* output, const T* input, const short* indices,
               int inputSize, int usedSize, int outputSize,
               hipStream_t stream) {
  // Each thread processes one input element
  // Only some of the threads (with valid mapping) write output
  const int kBlockSize = 256;
  const int kBlocks = DivUp(N * usedSize, kBlockSize);

  policyMap_kernel<T><<<kBlocks, kBlockSize, 0, stream>>>(
      (T*)output, (T*)input, (short*)indices, N, inputSize, usedSize,
      outputSize);
  ReportCUDAErrors(hipGetLastError());
}

template <typename T = float, bool use_se, ActivationFunction activation,
          bool use_bias, bool use_skip>
void OutputInputTransform(int N, int C, int se_K, T* output, const T* input,
                          const T* skip, const T* bias, const T* w1,
                          const T* b1, const T* w2, const T* b2,
                          hipStream_t stream) {
  // Each thread processes entire chess board
  if (use_se == false) {
    dim3 grid_dim(DivUp(C, kOpInpTransformBlockSize), N, 1);
    OutputTransform_relu_InputTransform_kernel<float, activation, use_bias, use_skip>
        <<<grid_dim, kOpInpTransformBlockSize, 0, stream>>>(N, C, output, input,
                                                            (float*)skip, bias);
  } else if (C > kMaxResBlockFusingChannels) {
    throw Exception(
        "res block fusing opt not supported for the given data type and no "
        "of filters\n");
  } else {
    OutputTransform_SE_relu_InputTransform_kernel<float, activation,
                                                  use_bias, use_skip>
        <<<N, C, 0, stream>>>(N, C, se_K, output, input, (float*)skip, bias, w1,
                              b1, w2, b2);
  }

  ReportCUDAErrors(hipGetLastError());
}

// N * C Tensors
// performs softmax along the C dimension
// Each thread processes one element
// Sums are computed in shared memory
// C threads per block, N blocks
template <typename T>
__global__ void softmax_kernel(T* output, const T* input) {
  int n = blockIdx.x;
  int c = threadIdx.x;
  int C = blockDim.x;
  int index = n * C + c;

  __shared__ float sum;
  if (c == 0) sum = 0;
  __syncthreads();

  // softmax = tf.exp(logits) / tf.reduce_sum(tf.exp(logits), axis)

  float x = (float)input[index];
  float ex = exp(x);

  // compute warp wide sums first
  float val = warpReduce(ex);

  // update shared memory sum across C dimension
  if ((c & 0x1F) == 0) atomicAdd(&sum, val);

  __syncthreads();

  float op = ex / sum;

  output[index] = (T)op;
}

template <typename T>
void Softmax(int N, int C, T* output, const T* input, hipStream_t stream) {
  softmax_kernel<T><<<N, C, 0, stream>>>(output, input);
  ReportCUDAErrors(hipGetLastError());
}

// N * C Tensors
// performs layer normalization along the C dimension
// Each thread processes one element
// Sums/variences are computed in shared memory
// C threads per block, N blocks
template <typename T>
__global__ void layer_norm_kernel(T* output, const T* input, const T* skip,
                                  const T* gammas, const T* betas, float ep) {
  int n = blockIdx.x;
  int c = threadIdx.x;
  int C = blockDim.x;

  __shared__ float sum, sum_sq;
  if (c == 0) {
    sum = 0;
    sum_sq = 0;
  }
  __syncthreads();

  int index = n * C + c;

  // From:
  // https://www.tensorflow.org/api_docs/python/tf/keras/layers/LayerNormalization
  // mean_i = sum(x_i[j] for j in range(k)) / k
  // var_i  = sum((x_i[j] - mean_i) ^ 2 for j in range(k)) / k
  // x_i_normalized = (x_i - mean_i) / sqrt(var_i + epsilon)
  // output_i = x_i_normalized * gamma + beta

  float x = (float)input[index];
  if (skip) x += (float)skip[index];

  float s = warpReduce(x);
  if ((c & 0x1F) == 0) atomicAdd(&sum, s);

  __syncthreads();

  float mean = sum / C;
  float d = x - mean;
  float d_sq = d * d;

  s = warpReduce(d_sq);
  if ((c & 0x1F) == 0) atomicAdd(&sum_sq, s);
  __syncthreads();

  float var = sum_sq / C;

  float norm = d / sqrt(var + ep);
  float op = norm * (float)gammas[c] + (float)betas[c];

  output[index] = (T)op;
}

// add (optional) skip connection to input, and then perform Layer normalization
// normalization is done across C dimension (i.e, sums and std deviations taken
// over elements in C dim)
template <typename T>
void LayerNorm(int N, int C, T* output, const T* input, const T* skip,
               const T* gammas, const T* betas, float ep, hipStream_t stream) {
  layer_norm_kernel<T>
      <<<N, C, 0, stream>>>(output, input, skip, gammas, betas, ep);
  ReportCUDAErrors(hipGetLastError());
}

// Compute promotion logits in a single kernel
// keys matrix is of N * 64 * C (but we use only last 8 from the 'rows'
// dimension, so N * 8 * C)
// ppo matrix is 4 * C (weights for dense layer / matrix multiplication)
// policy_attn_logits matrix is N * 64 * 64, but we use only 8x8 part of it
// from each batch dimension (so, N * 8 * 8)
// output matrix (promotion logits) is of N * 8 * 24 size
template <typename T>
__global__ void promotion_logits_kernel(int C, T* output, const T* keys,
                                        const T* ppo,
                                        const T* policy_attn_logits) {
  constexpr int output_stride = 64 * 64 + 8 * 24;
  int n = blockIdx.x;   // [0..N)
  int y = threadIdx.y;  // [0..8)
  int x = threadIdx.x;  // [0..24)     // Can split into 8 * 3

  int threadInGroup = threadIdx.y * 24 + threadIdx.x;

  // phase 1 : compute promotion_offsets by multiplying keys and ppo matrices
  const T* keys_start =
      keys + n * 64 * C + C * 56;  // we are interested only in last 8 out of 64
                                   // 'rows' of keys matrix
  __shared__ float promotion_offsets[4][8];

  // only 32 threads out of 192 in the group are active in this phase, and each
  // thread computes one element of the promotion_offsets matrix
  // TODO: opt idea1, can use more threads to reduce the length of the loop for
  // the matrix multiply (do parallel reduction of partial sums later)
  //       opt idea2, the below loop for matrix mul has very poor memory access
  //       pattern, can do the loop over 32, and do parallel reductions
  if (threadInGroup < 32) {
    int x = threadInGroup % 4;
    int y = threadInGroup / 4;

    float S = 0;
    for (int i = 0; i < C;
         i++) {  // TODO: modify to loop over 32 instead of C (doing parallel
                 // reductions for the 32 sums)
      float a = (float)keys_start[y * C + i];
      float b =
          (float)ppo[x * C + i];  // weight matrix is transposed (col major)
      S += a * b;
    }

    // write the product (promotion_offsets) in shared memory
    promotion_offsets[x][y] = S;
  }

  __syncthreads();

  // phase 2: add the last "row" to the other 3
  // #knight offset is added to the other three
  // promotion_offsets = promotion_offsets[:, :3, :] + promotion_offsets[:, 3:4,
  // :] 
  // Only 24 threads in the group are active in this phase
  if (threadInGroup < 32) {
    int x = threadInGroup % 4;
    int y = threadInGroup / 4;
    if (x < 3) {
      promotion_offsets[x][y] += promotion_offsets[3][y];
    }
  }

  __syncthreads();

  // phase 3: add 8x8 chunk of policy_attn_logits matrix to promotion offsets
  //          the output is 3x8x8 (written as 8 * 24)
  // All threads are active in this phase and they compute one element each
  int w = x / 3;
  int c = x % 3;

  // n_promo_logits = matmul_qk[:, -16:-8, -8:]  # default traversals from rank
  // 7 to rank 8
  float n_promo_logit =
      (float)policy_attn_logits[n * output_stride + (48 + y) * 64 + (56 + w)];
  float promo_offset = promotion_offsets[c][w];

  float op = n_promo_logit + promo_offset;

  output[n * output_stride + threadInGroup] = (T)op;
}

template <typename T>
void ComputePromotionLogits(int N, int C, T* output, const T* keys,
                            const T* ppo, const T* policy_attn_logits,
                            hipStream_t stream) {
  // N blocks
  // 8 * 24 threads
  // Each thread computes a single output element
  dim3 blockDim(24, 8, 1);
  promotion_logits_kernel<T>
      <<<N, blockDim, 0, stream>>>(C, output, keys, ppo, policy_attn_logits);
}

// Template instantiation.
template void copyTypeConverted<half, float>(half* op, float* ip, int N,
                                             hipStream_t stream);
template void copyTypeConverted<float, half>(float* op, half* ip, int N,
                                             hipStream_t stream);
template void copyTypeConverted<float, float>(float* op, float* ip, int N,
                                              hipStream_t stream);
template void copyTypeConverted<half, half>(half* op, half* ip, int N,
                                            hipStream_t stream);

template void batchNorm<float>(float* output, const float* input,
                               const float* skipInput, int N, int C, int H,
                               int W, float* means, float* var_multipliers,
                               ActivationFunction activation);
template void batchNorm<half>(half* output, const half* input,
                              const half* skipInput, int N, int C, int H, int W,
                              float* means, float* var_multipliers,
                              ActivationFunction activation);

template void addVectors<float>(float* c, float* a, float* b, int size,
                                int asize, int bsize, ActivationFunction act,
                                hipStream_t stream);
template void addVectors<half>(half* c, half* a, half* b, int size, int asize,
                               int bsize, ActivationFunction act,
                               hipStream_t stream);

template void addBias_NCHW<float>(float* c, float* a, float* b, int N, int C,
                                  int H, int W, ActivationFunction activation,
                                  hipStream_t stream);

template void addBias_NCHW<half>(half* c, half* a, half* b, int N, int C, int H,
                                 int W, ActivationFunction activation,
                                 hipStream_t stream);

template void globalAvgPool<float>(int N, int C, float* output,
                                   const float* input,
                                   const float* prevLayerBias, bool nhwc);
template void globalAvgPool<half>(int N, int C, half* output, const half* input,
                                  const half* prevLayerBias, bool nhwc);

template void globalScale<float>(int N, int C, float* output,
                                 const float* input, const float* scaleBias,
                                 const float* prevLayerBias, bool nhwc,
                                 ActivationFunction activation);
template void globalScale<half>(int N, int C, half* output, const half* input,
                                const half* scaleBias,
                                const half* prevLayerBias, bool nhwc,
                                ActivationFunction activation);

template void PolicyMap<float>(int N, float* output, const float* input,
                               const short* indices, int inputSize,
                               int usedSize, int outputSize,
                               hipStream_t stream);

template void PolicyMap<half>(int N, half* output, const half* input,
                              const short* indices, int inputSize, int usedSize,
                              int outputSize, hipStream_t stream);

template void FilterTransform<float>(int N, int C, float* transformedFilter,
                                     const float* filter);

template void InputTransform<float, true>(int N, int C,
                                          float* transformed_input,
                                          const float* input,
                                          hipStream_t stream);

template void InputTransform<float, false>(int N, int C,
                                           float* transformed_input,
                                           const float* input,
                                           hipStream_t stream);

template void OutputTransform<float, true, RELU, true, true, false, false>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputTransform<float, false, RELU, true, true, false, false>(

    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputTransform<float, true, RELU, true, true, true, false>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputTransform<float, false, RELU, true, true, true, false>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputTransform<float, false, RELU, true, false, false, false>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputTransform<float, false, RELU, true, false, false, true>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputTransform<float, true, RELU, true, true, true, true>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputTransform<float, true, MISH, true, true, false, false>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputTransform<float, false, MISH, true, true, false, false>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputTransform<float, true, MISH, true, true, true, false>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputTransform<float, false, MISH, true, true, true, false>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputTransform<float, false, MISH, true, false, false, false>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputTransform<float, false, MISH, true, false, false, true>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputTransform<float, true, MISH, true, true, true, true>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputTransform<float, false, NONE, true, false, false, false>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputInputTransform<float, true, RELU, true, true>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputInputTransform<float, false, RELU, true, true>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputInputTransform<float, false, RELU, true, false>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputInputTransform<float, true, MISH, true, true>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputInputTransform<float, false, MISH, true, true>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void OutputInputTransform<float, false, MISH, true, false>(
    int N, int C, int se_K, float* output, const float* input,
    const float* skip, const float* bias, const float* w1, const float* b1,
    const float* w2, const float* b2, hipStream_t stream);

template void Softmax<half>(int N, int C, half* output, const half* input,
                            hipStream_t stream);
template void Softmax<float>(int N, int C, float* output, const float* input,
                             hipStream_t stream);

template void LayerNorm<half>(int N, int C, half* output, const half* input,
                              const half* skip, const half* gammas,
                              const half* betas, float ep, hipStream_t stream);
template void LayerNorm<float>(int N, int C, float* output, const float* input,
                               const float* skip, const float* gammas,
                               const float* betas, float ep,
                               hipStream_t stream);

template void ComputePromotionLogits<half>(int N, int C, half* output,
                                           const half* keys, const half* ppo,
                                           const half* policy_attn_logits,
                                           hipStream_t stream);
template void ComputePromotionLogits<float>(int N, int C, float* output,
                                            const float* keys, const float* ppo,
                                            const float* policy_attn_logits,
                                            hipStream_t stream);

template void convertNCHWtoNHWC<half, float>(half* output_tensor,
                                             const float* input_tensor, int Nin,
                                             int Cin, int Nout, int Cout, int H,
                                             int W);
template void convertNCHWtoNHWC<float, float>(float* output_tensor,
                                              const float* input_tensor,
                                              int Nin, int Cin, int Nout,
                                              int Cout, int H, int W);
template void convertNCHWtoNHWC<half, half>(half* output_tensor,
                                            const half* input_tensor, int Nin,
                                            int Cin, int Nout, int Cout, int H,
                                            int W);
}  // namespace cudnn_backend
}  // namespace lczero
